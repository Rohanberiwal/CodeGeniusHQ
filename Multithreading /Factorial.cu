#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

const int N = 10;  // Factorial of N

// CUDA kernel to calculate factorial
__global__ void factorialKernel(int *result) {
    int tid = threadIdx.x;

    if (tid == 0) {
        *result = 1;  // Initialize result
    }

    __syncthreads();

    for (int i = tid + 1; i <= N; i += blockDim.x) {
        atomicMul(result, i);  // Multiply result by i
    }
}

int main() {
    int result;
    int *d_result;

    // Allocate device memory for result
    hipMalloc((void**)&d_result, sizeof(int));

    // Launch kernel with one block and as many threads as N
    factorialKernel<<<1, N>>>(d_result);

    // Copy result from device to host
    hipMemcpy(&result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    // Print the result
    std::cout << "Factorial of " << N << " is: " << result << std::endl;

    // Free device memory
    hipFree(d_result);

    return 0;
}
